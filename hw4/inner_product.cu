
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>
#include <time.h>
#include <stdlib.h>

void reduction(double* sum_ptr, const double* a, const double* b, long M, long N){
  double sum = 0;
  for (long j=0; j < M; j++) {
    sum=0;
  #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < N; i++){
    sum += a[j*M+i]*b[i];
  }
  sum_ptr[j]=sum;
}
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

#define BLOCK_SIZE 1024

// Warp divergence
__global__ void reduction_kernel0(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x %   2 == 0) smem[threadIdx.x] += smem[threadIdx.x + 1];
  __syncthreads();
  if (threadIdx.x %   4 == 0) smem[threadIdx.x] += smem[threadIdx.x + 2];
  __syncthreads();
  if (threadIdx.x %   8 == 0) smem[threadIdx.x] += smem[threadIdx.x + 4];
  __syncthreads();
  if (threadIdx.x %  16 == 0) smem[threadIdx.x] += smem[threadIdx.x + 8];
  __syncthreads();
  if (threadIdx.x %  32 == 0) smem[threadIdx.x] += smem[threadIdx.x + 16];
  __syncthreads();
  if (threadIdx.x %  64 == 0) smem[threadIdx.x] += smem[threadIdx.x + 32];
  __syncthreads();
  if (threadIdx.x % 128 == 0) smem[threadIdx.x] += smem[threadIdx.x + 64];
  __syncthreads();
  if (threadIdx.x % 256 == 0) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x % 512 == 0) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x == 0) sum[blockIdx.x] = smem[threadIdx.x] + smem[threadIdx.x + 512];
}

// Shared memory bank conflicts
__global__ void reduction_kernel1(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x *   2] += smem[threadIdx.x *   2 +   1];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x *   4] += smem[threadIdx.x *   4 +   2];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x *   8] += smem[threadIdx.x *   8 +   4];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x *  16] += smem[threadIdx.x *  16 +   8];
  __syncthreads();
  if (threadIdx.x <  32) smem[threadIdx.x *  32] += smem[threadIdx.x *  32 +  16];
  __syncwarp();
  if (threadIdx.x <  16) smem[threadIdx.x *  64] += smem[threadIdx.x *  64 +  32];
  __syncwarp();
  if (threadIdx.x <   8) smem[threadIdx.x * 128] += smem[threadIdx.x * 128 +  64];
  __syncwarp();
  if (threadIdx.x <   4) smem[threadIdx.x * 256] += smem[threadIdx.x * 256 + 128];
  __syncwarp();
  if (threadIdx.x <   2) smem[threadIdx.x * 512] += smem[threadIdx.x * 512 + 256];
  __syncwarp();
  if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[512];
}

__global__ void reduction_product(double* sum, const double* a, const double* b, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx]*b[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void reduction_kernel2(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

int main() {
  long N = (1UL<<24);
  // long M = (1UL<<10);
  srand((int)time(0));
  double *a, *b;
  hipHostMalloc((void**)&a, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&b, N * sizeof(double), hipHostMallocDefault);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++){
    a[i] = ((double)rand())/RAND_MAX;
    b[i] = ((double)rand())/RAND_MAX;
  }

  double sum_ref, sum;
  double tt = omp_get_wtime();
  reduction(&sum_ref, a, b, 1, N);
  printf("CPU Bandwidth = %f GB/s\n", 2*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double *a_d, *b_d, *y_d;
  hipMalloc(&a_d, N*sizeof(double));
  hipMalloc(&b_d, N*sizeof(double));
  long N_work = 1;
  for (long i = (N+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) N_work += i;
  hipMalloc(&y_d, N_work*sizeof(double)); // extra memory buffer for reduction across thread-blocks
  hipMemcpyAsync(a_d, a, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(b_d, b, N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  tt = omp_get_wtime();


  double* sum_d = y_d;
  long Nb = (N+BLOCK_SIZE-1)/(BLOCK_SIZE);
  reduction_product<<<Nb,BLOCK_SIZE>>>(sum_d, a_d, b_d, N);
  while (Nb > 1) {
    long N1 = Nb;
    Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
    reduction_kernel2<<<Nb,BLOCK_SIZE>>>(sum_d + N1, sum_d, N1);
    sum_d += N1;
  }


  hipMemcpyAsync(&sum, sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 2*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  printf("Error = %f\n", fabs(sum-sum_ref));

  hipFree(a_d);
  hipFree(b_d);
  hipFree(y_d);

  return 0;
}
